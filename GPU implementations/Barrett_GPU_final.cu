/*

Use this command to execute in Google Collab
!nvcc -arch=sm_75 Barrett_GPU.cu -o Barrett_GPU
!./Barrett_GPU

*/

#include <stdio.h>    
#include <stdlib.h>   
#include <string.h>   
#include <hip/hip_runtime.h>
#include <stdint.h>

#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        exit(EXIT_FAILURE); \
    } \
}
//required to know if there are no kernel issues


// TIMERS FOR BENCHMARKING
void startTimer(hipEvent_t* start, hipEvent_t* stop) {
    CHECK_CUDA_ERROR(hipEventCreate(start)); 
    CHECK_CUDA_ERROR(hipEventCreate(stop));
    CHECK_CUDA_ERROR(hipEventRecord(*start));
}


float stopTimer(hipEvent_t start, hipEvent_t stop) {
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop)); 
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    return milliseconds;
}

// DEVICE INFORMATION
void deviceinfo() {
    hipDeviceProp_t prop; 
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0)); 

    size_t free_memory, total_memory; 
    CHECK_CUDA_ERROR(hipMemGetInfo(&free_memory, &total_memory)); 
    
    printf("GPU: %s\n", prop.name); 
  // printf("Compute : %d.%d\n", prop.major, prop.minor); 
   // printf(" GPU Mem: %.2f GB\n", free_memory / (1024.0 * 1024.0 * 1024.0));

}

// PRINT U64 
//prints needed to debug
void print_u64(const char* label, uint64_t value) {
    printf("%s: %llu (0x%llx)\n", label, value, value); 
}



// MU CALCULATION
uint64_t compute_barrett_mu(uint64_t n) {
    if (n == 0) { 
        fprintf(stderr, "Error: Moduluscannot be zero\n"); 
        exit(EXIT_FAILURE); 
    }
    if (n == 1) {
        return 0xFFFFFFFFFFFFFFFFULL; 
    }

    uint64_t mu_val = (uint64_t)((((__uint128_t)-1) / n) + 1);
    return mu_val; 
}

// BARRET REDUCTION FUNCTION
__device__ uint64_t barrett_reduce(uint64_t x_low, uint64_t x_high, uint64_t n, uint64_t mu) {

    __uint128_t X = ((__uint128_t)x_high << 64) | x_low;
    uint64_t tmp1_high_xl_mu = __umul64hi(x_low, mu);
    
    uint64_t tmp2_high_xh_mu = __umul64hi(x_high, mu);
    uint64_t tmp2_low_xh_mu = x_high * mu;

    uint64_t sum_mid_part = tmp2_low_xh_mu + tmp1_high_xl_mu;
    uint64_t carry_to_q = (sum_mid_part < tmp2_low_xh_mu); 

    uint64_t q_val = tmp2_high_xh_mu + carry_to_q;

    __uint128_t r = X - ((__uint128_t)q_val * n);

    while (r >= n) {
        r -= n;
    }

    return (uint64_t)r; 
}
__device__ uint64_t mod_mul(uint64_t a, uint64_t b, uint64_t n, uint64_t mu) {
    __uint128_t product = (__uint128_t)a * b;
    return barrett_reduce((uint64_t)product, (uint64_t)(product >> 64), n, mu);
}
// BARRET MODULAR EXPONENTIATION
__device__ uint64_t modexp_barrett(uint64_t base, uint64_t exp, uint64_t n, uint64_t mu) {
    uint64_t result = 1; 
    base = base % n;     

    while (exp > 0) { 
        if (exp & 1) { 
            //result = barrett_reduce((uint64_t)prod, (uint64_t)(prod >> 64), n, mu);
            result = mod_mul(result, base, n, mu);

        }
        __uint128_t prod = (__uint128_t)base * base; 
        base = barrett_reduce((uint64_t)prod, (uint64_t)(prod >> 64), n, mu);
        exp >>= 1;
    }
    return result; 
}


// RSA LAUNCH FUNCTION
__global__ void rsa_barrett_kernel(uint64_t* out, uint64_t base, uint64_t exp, uint64_t n, uint64_t mu) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        out[0] = modexp_barrett(base, exp, n, mu); 
    }
}

int main() {
    deviceinfo(); 

    uint64_t p = 61;
    uint64_t q = 53;
    uint64_t n = p * q;        
    uint64_t e = 17;           
    uint64_t d = 2753;         
    uint64_t msg = 123;        

    uint64_t mu = compute_barrett_mu(n); 

    printf("\nRSA Parameters:\n");
    print_u64("  Modulus (n)", n);
    print_u64("  Public Key (e)", e);
    print_u64("  Private Key (d)", d);
    print_u64("  Message", msg);
    print_u64("  Computed Mu", mu);

    uint64_t* d_result; 
    CHECK_CUDA_ERROR(hipMalloc(&d_result, sizeof(uint64_t))); 

    hipEvent_t start, stop;
    float total_encryption_time_ms = 0; 
    float total_decryption_time_ms = 0; 

    int num_iterations = 1000; 

    printf("\n === Running Benchmarking (%d Iterations) ===\n\n", num_iterations);

    for (int i = 0; i < num_iterations; ++i) { 
        startTimer(&start, &stop); 
        rsa_barrett_kernel<<<1, 1>>>(d_result, msg, e, n, mu); 
        uint64_t cipher;
        CHECK_CUDA_ERROR(hipMemcpy(&cipher, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost)); 
        total_encryption_time_ms += stopTimer(start, stop); 

        startTimer(&start, &stop); 
        rsa_barrett_kernel<<<1, 1>>>(d_result, cipher, d, n, mu); 
        uint64_t decrypted;
        CHECK_CUDA_ERROR(hipMemcpy(&decrypted, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost));
        total_decryption_time_ms += stopTimer(start, stop);

        // ERROR VERIFICATION
        int err_cntr = 0;
        if (decrypted != msg) { 
            printf("\nVerification: FAILED! Decrypted message (%llu) does NOT match original (%llu).\n", decrypted, msg);
            ++err_cntr;
        }
        if(err_cntr > 0 ) {
            printf("\nVerification failed %d times.\n", err_cntr);
        }
        else if((err_cntr == 0) & (i == num_iterations - 1) ) {
            printf("\nAll %d iterations PASSED!\n", num_iterations);
        }


        // PRINT ONLY ON THE LAST ITERATION
        if (i == num_iterations - 1) {
            printf("\n--- Final Verification (from last iteration) ---\n");
            printf("Encrypted Result: %llu\n", cipher);
            printf("Decrypted Result: %llu\n", decrypted);
            if (decrypted == msg) { 
                printf("Verification: SUCCESS! Decrypted message matches original.\n");
            } else {
                printf("Verification: FAILED! Decrypted message (%llu) does NOT match original (%llu).\n", decrypted, msg);
            }
        }
    }

    printf("\n--- Benchmarking Summary ---\n");
    printf("Average Encryption Time: %.4f ms\n", total_encryption_time_ms / num_iterations);
    printf("Average Decryption Time: %.4f ms\n", total_decryption_time_ms / num_iterations);

    CHECK_CUDA_ERROR(hipFree(d_result));
    return 0; 
}
